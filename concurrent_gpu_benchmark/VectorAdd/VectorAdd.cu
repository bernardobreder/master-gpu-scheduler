
#include <hip/hip_runtime.h>


__global__ void addKernel01(int *c, int *a, int *b, int repeat)
{
	__shared__ unsigned char s[48 * 1024];
	int i = threadIdx.x;
	int j = i;
	for (int n = 0; n < repeat; n++) s[i % 64] = 1;
	for (int n = 0; n < repeat; n++) c[j] = a[i] + b[i] + s[i % 64];
}

__global__ void addKernel02(int *c, int *a, int *b, int repeat)
{
	__shared__ unsigned char s[48 * 1024];
	int i = threadIdx.x;
	int j = i;
	for (int n = 0; n < repeat; n++) s[i % 64] = 1;
	for (int n = 0; n < repeat; n++) c[j] = a[i] + b[i] + s[i % 64];
}

__global__ void addKernel03(int *c, int *a, int *b, int repeat)
{
	__shared__ unsigned char s[48 * 1024];
	int i = threadIdx.x;
	int j = i;
	for (int n = 0; n < repeat; n++) s[i % 64] = 1;
	for (int n = 0; n < repeat; n++) c[j] = a[i] + b[i] + s[i % 64];
}

__global__ void addKernel04(int *c, int *a, int *b, int repeat)
{
	__shared__ unsigned char s[48 * 1024];
	int i = threadIdx.x;
	int j = i;
	for (int n = 0; n < repeat; n++) s[i % 64] = 1;
	for (int n = 0; n < repeat; n++) c[j] = a[i] + b[i] + s[i % 64];
}

__global__ void addKernel05(int *c, int *a, int *b, int repeat)
{
	__shared__ unsigned char s[48 * 1024];
	int i = threadIdx.x;
	for (int n = 0; n < repeat; n++) s[i % 64] = 1;
	for (int n = 0; n < repeat; n++) c[i] = a[i] + b[i] + s[i % 64];
}

__global__ void addKernel06(int *c, int *a, int *b, int repeat)
{
	__shared__ unsigned char s[48 * 1024];
	int i = threadIdx.x;
	int j = i;
	for (int n = 0; n < repeat; n++) s[i % 64] = 1;
	for (int n = 0; n < repeat; n++) c[j] = a[i] + b[i] + s[i % 64];
}

__global__ void addKernel07(int *c, int *a, int *b, int repeat)
{
	__shared__ unsigned char s[48 * 1024];
	int i = threadIdx.x;
	int j = i;
	for (int n = 0; n < repeat; n++) s[i % 64] = 1;
	for (int n = 0; n < repeat; n++) c[j] = a[i] + b[i] + s[i % 64];
}

__global__ void addKernel08(int *c, int *a, int *b, int repeat)
{
	__shared__ unsigned char s[32 * 1024];
	int i = threadIdx.x;
	int j = i;
	for (int n = 0; n < repeat; n++) s[i % 64] = 1;
	for (int n = 0; n < repeat; n++) c[j] = a[i] + b[i] + s[i % 64];
}

__global__ void addKernel09(int *c, int *a, int *b, int repeat)
{
	__shared__ unsigned char s[40 * 1024];
	int i = threadIdx.x;
	int j = i;
	for (int n = 0; n < repeat; n++) s[i % 64] = 1;
	for (int n = 0; n < repeat; n++) c[j] = a[i] + b[i] + s[i % 64];
}

__global__ void addKernel10(int *c, int *a, int *b, int repeat)
{
	__shared__ unsigned char s[16 * 1024];
	int i = threadIdx.x;
	int j = i;
	for (int n = 0; n < repeat; n++) s[i % 64] = 1;
	for (int n = 0; n < repeat; n++) c[j] = a[i] + b[i] + s[i % 64];
}

__global__ void addKernel11(int *c, int *a, int *b, int repeat)
{
	__shared__ unsigned char s[8 * 1024];
	int i = threadIdx.x;
	int j = i;
	for (int n = 0; n < repeat; n++) s[i % 64] = 1;
	for (int n = 0; n < repeat; n++) c[j] = a[i] + b[i] + s[i % 64];
}